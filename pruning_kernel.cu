#include "hip/hip_runtime.h"
#ifndef __HIPCC__  
    #define __HIPCC__
#endif

#define MALLOC_CHECK_ 2

#include <stdio.h>
#include <limits.h>
#include "pruning.h"

/*
    Alpha-Beta pruning for the game Reversi/Othello.
    Simulation program, not designed to run a full game.
*/

__device__
int directions[9][2] = {{-1,-1}, {-1, 0}, {-1, 1},
                        { 0,-1}, { 0, 0}, { 0, 1},
                        { 1,-1}, { 1, 0}, { 1, 1}};

__device__ int *galpha, *gbeta;


// From cuda zero-sum games presentation
__device__
void resolve(int *alpha, int *beta) {
    if (*alpha <= *galpha) *alpha = *galpha;
    else atomicMax(galpha, *alpha);

    if (*beta >= *gbeta) *beta = *gbeta;
    else atomicMin(gbeta, *beta);
}

// Check if direction results in valid placement
__device__
bool check_valid(board_t *board, state_t state, int dir, int x, int y) {
    int cur_x = x + directions[dir][0];
    int cur_y = y + directions[dir][1];

    while (BOUND(board->dim_x, cur_x) && BOUND(board->dim_y, cur_y)) {

        state_t cur_state = board->states[cur_x * board->dim_y + cur_y];

        if (cur_state == EMPTY) {
            return true;
        }
        if (cur_state == state) {
            return false;
        }

        cur_x += directions[dir][0];
        cur_y += directions[dir][1];
    }

    return false;
}

// Check if [x,y] is a valid new move
__device__
void valid_move(board_t *board, state_t state, int x, int y) {

    state_t cur_state = board->states[x * board->dim_y + y];
    if (cur_state != EMPTY) {
        return;
    }

    // for (int i = MAX(x - 1, 0); i <= MIN(x + 1, board->dim_x); i++) {
    //     for (int j = MAX(y - 1, 0); j <= MIN(y + 1, board->dim_y); j++) {

    //         if (i == j) continue;
    //         state_t new_state = board->states[i * board->dim_y + j];

    //         if (new_state != EMPTY && new_state != state) {
    //             int dir = (i + 1) * 3 + (j + 1);
    //             if (check_valid(board, state, dir, i, j)) {
    //                 board->states[x * board->dim_y + y] = VALID;
    //             }
    //         }
    //     }
    // }

    for (int k = 0; k < 9; k++) {
        if (k == 4) continue;

        int i = x + directions[k][0];
        int j = y + directions[k][1];

        if (!BOUND(board->dim_x, i) || !BOUND(board->dim_y, j)) {
            continue;
        }

        state_t new_state = board->states[i * board->dim_y + j];
        if (new_state != EMPTY && new_state != state) {
            if (check_valid(board, state, k, i, j)) {
                board->states[x * board->dim_y + y] = VALID;
            }
        }

    }
}

// Check how much position [x,y] contributes to heuristic
// Simplistic for project to avoid divergent behavior; more powerful ones available.
__device__
int local_eval(board_t *board, state_t state, int x, int y) {
    if (state == board->states[x * board->dim_y + y]) {
        return 1;
    }
    return 0;
}

// Orchestrate prefix-sum (reduce) for final evalution value
__device__
int eval_function(board_t *board, state_t state, int x, int y, int *shared) {

    unsigned int tid = threadIdx.x * blockDim.x + threadIdx.y;
    shared[tid] = local_eval(board, state, x, y);
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            shared[tid] += shared[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        return shared[0];
    }
}

// Fail-soft alpha-beta pruning adapted from wikipedia
__device__
int node_traverse(board_t *board, int depth, int alpha, int beta, state_t state, int *sdata) {

    int x = threadIdx.x, y = threadIdx.y, value = 0, count = 0;

    if (depth == 0) {
        return eval_function(board, state, x, y, (int *) sdata);
    }

    // Mark valid moves
    valid_move(board, state, x, y);
    __syncthreads();

    if (state == BLACK) {
        value = INT_MIN;

        // Iterate through all moves
        count = 0;
        for (int i = 0; i < board->dim_x; i++) {
            for (int j = 0; j < board->dim_y; j++) {

                if (count >= MAX_MOVES) {
                    goto end;
                }

                if (board->states[i] != VALID) continue;
                if (x == i && y == j) {
                    board->states[i] = state;
                }

                // Maximize score!
                value = MAX(value, node_traverse(board, depth - 1, alpha, beta, WHITE, sdata));
                if (value > beta) {
                    goto end; // β cutoff
                }

                if (threadIdx.x == 0 && threadIdx.y == 0) {
                    alpha = MAX(alpha, value);
                    resolve(&alpha, &beta);
                }

                // Reverse move
                if (x == i && y == j) {
                    board->states[i] = state;
                }

                count += 1;
            }
        }

    } else if (state == WHITE) {
        value = INT_MAX;

        // Iterate through all moves
        count = 0;
        for (int i = 0; i < board->dim_x; i++) {
            for (int j = 0; j < board->dim_y; j++) {

                if (count >= MAX_MOVES) {
                    goto end;
                }

                if (board->states[i] != VALID) continue;
                if (x == i && y == j) {
                    board->states[i] = state;
                }

                // Minimize score!
                value = MIN(value, node_traverse(board, depth - 1, alpha, beta, BLACK, sdata));
                if (value < alpha) {
                    goto end; // α cutoff
                }

                if (threadIdx.x == 0 && threadIdx.y == 0) {
                    beta = MIN(beta, value);
                    resolve(&alpha, &beta);
                }

                // Reverse move
                if (x == i && y == j) {
                    board->states[i] = state;
                }

                count += 1;
            }
        }
    }
    end: return value;
}

__global__
void traverse_wrapper(board_t *board, int *depth) {

    extern __shared__ int sdata[];
    // int idx = blockIdx.x;
    node_traverse(board, *depth, INT_MIN, INT_MAX, BLACK, (int *) sdata);
}

float parallel_prune(int depth, int dim_x, int dim_y) {
    int *d_depth;
    state_t *d_states;
    board_t *d_board, board;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void **) &d_depth, sizeof(int));
    hipMemcpy(d_depth, &depth, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **) &d_states, sizeof(state_t) * dim_x * dim_y);
    state_t *base_states = setup_board(dim_x, dim_y);
    hipMemcpy(d_states, base_states, sizeof(state_t) * dim_x * dim_y, hipMemcpyHostToDevice);

    board.dim_x = dim_x;
    board.dim_y = dim_y;
    board.states = d_states;

    hipMalloc((void **) &d_board, sizeof(board_t));
    hipMemcpy(d_board, &board, sizeof(board_t), hipMemcpyHostToDevice);

    int alpha = INT_MIN;
    int beta = INT_MAX;

    hipMemcpyToSymbol(HIP_SYMBOL("galpha"), &alpha, sizeof(int), size_t(0), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("gbeta"), &beta, sizeof(int), size_t(0), hipMemcpyHostToDevice);

    dim3 grid(1);
    dim3 block(dim_x, dim_y);

    hipEventRecord(start);
    traverse_wrapper <<<grid, block, dim_x * dim_y>>> (d_board, d_depth);
    hipEventRecord(stop);

    hipFree(d_depth);
    hipFree(d_states);
    hipFree(d_board);

    hipEventSynchronize(stop);
    float milliseconds = 0.0;
    hipEventElapsedTime(&milliseconds, start, stop);

    return milliseconds;

    // hipMalloc((void **) &d_boards, sizeof(state_t *) * MAX_MOVES);

    // for (int i = 0; i < depth; i++) {
    //     hipMemcpy(&(d_depths[i]), &depth, sizeof(int), hipMemcpyHostToDevice);
    // }

    // state_t *base_board = setup_board(dim_x, dim_y);

    // for (int i = 0; i < MAX_MOVES; i++) {
    //     state_t *d_board;
    //     hipMalloc((void **) &d_board, sizeof(state_t) * dim_x * dim_y);
    //     boards[i] = d_board;
    //     // hipMemcpy(d_board, base_board, sizeof(state_t) * dim_x * dim_y, hipMemcpyHostToDevice);
    //     hipMemcpy(&(d_boards[i]), &d_board, sizeof(state_t *), hipMemcpyHostToDevice);
    // }

    // int cur_move[2] = {(dim_x / 2) - 1, (dim_y / 2) + 1}; // Arbitrary leftmost move, heurestic used for better performance
    // state_t m_states[2] = {BLACK, WHITE};

    // state_t cur_state = m_states[i % 2];

    // Setup boards
    // for (int j = 0; j < MAX_MOVES; j++) {
    //     hipMemcpy(boards[j], base_board, sizeof(state_t) * dim_x * dim_y, hipMemcpyHostToDevice);
    // }

    // // Perform leftmost move of game tree
    // base_board[cur_move[0] * dim_y + cur_move[1]] = cur_state;

    // // Update move
    // if (cur_move[1] < dim_y) {
    //     cur_move[1] += 1;
    // } else {
    //     cur_move[0] += 1;
    // }

    // // Get evaluation
    // int val = seq_eval(base_board, dim_x, dim_y, cur_state);
    // if (i % 2 == 0) {
    //     alpha = max(val, alpha);
    // } else {
    //     beta = min(val, beta);
    // }
}
